#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>
#include "KMParams.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "km_cuda.cuh"
#include "point.h"

__global__ void test_cuda(const KMParams *kmp, const point_data_t *data,
                          point_data_t *centroids) {
    for (int i = 0; i < kmp->clusters; ++i) {
        for (int j = 0; j < kmp->dim; ++j) {
            centroids[i * kmp->dim + j] = 777;
        }
    }
}

void km_cuda_run(const KMParams &host_kmp, const point_data_t *host_data,
                 point_data_t *host_centroids, std::chrono::duration<double> &t,
                 unsigned &iterations) {
    // copy params to device
    KMParams *kmp;
    hipMalloc((void **)&kmp, sizeof(KMParams));
    hipMemcpy(kmp, &host_kmp, sizeof(KMParams), hipMemcpyHostToDevice);

    // copy points to device
    const unsigned data_size = host_kmp.n * host_kmp.dim * sizeof(point_data_t);
    point_data_t *data;
    hipMalloc(&data, data_size);
    hipMemcpy(data, host_data, data_size, hipMemcpyHostToDevice);

    // copy centroids to device
    const unsigned centroids_size =
        host_kmp.clusters * host_kmp.dim * sizeof(point_data_t);
    point_data_t *centroids;
    hipMalloc(&centroids, centroids_size);
    hipMemcpy(centroids, host_centroids, centroids_size,
               hipMemcpyHostToDevice);

    // run kernel
    test_cuda<<<1, 1>>>(kmp, data, centroids);
    // hipDeviceSynchronize();

    std::cout << "Ran cuda kernel." << std::endl;

    // copy centroids back
    hipMemcpy(host_centroids, centroids, centroids_size,
               hipMemcpyDeviceToHost);

    hipFree(centroids);
    hipFree(data);
}
